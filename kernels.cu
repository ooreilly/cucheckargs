
#include <hip/hip_runtime.h>
extern "C" __global__ void add(float *a, float *b) { 
}

// Strange, but valid formatting
        extern "C"     __global__    void       add2(
                        float *  a, 
                        float *b) { 
}

// This function gets called with too few arguments
extern "C" __global__ void toofew(float *a, float *b) { 
}

// This function gets called with too many arguments
extern "C" __global__ void toomany(float *a, float *b, int *c) { 
}

// Extern block containing several functions
extern "C" {
        __global__ void block_toofew(float *a, float *b) { }
        __global__ void block_toomany(float *a, float *b, int *c) {  }
}
